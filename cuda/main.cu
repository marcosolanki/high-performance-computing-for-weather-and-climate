#include "device.cuh"
#include "host.cuh"

#include <chrono>
#include <fstream>
#include <sstream>

#ifndef CUDART_VERSION
#error "CUDART_VERSION not defined!"
#endif


using time_point = std::chrono::time_point<std::chrono::steady_clock>;


// run_simulation<T>():
// Runs the 4th-order diffusion simulation on a CUDA-enabled GPU and writes its results to some output files.
//
// Input:   xsize, ysize, zsize :: Dimensions of the domain (including boundary points)
//          itrs                :: Number of timestep iterations
//          bdry                :: Number of boundary points (halo size)
//          mode                :: Computation mode (double-5/13-point stencil(s) with/without shared memory)
//          T                   :: Numeric real type
// Output:  return (...)        :: Measured time (memory transfer + device allocation + computation) in seconds
template<typename T>
double run_simulation(std::size_t xsize, std::size_t ysize, std::size_t zsize, std::size_t itrs, std::size_t bdry, Mode mode) {

    constexpr T alpha = static_cast<T>(1) / 32;
    const std::size_t xmin = bdry, xmax = xsize - bdry;
    const std::size_t ymin = bdry, ymax = ysize - bdry;
    const std::size_t zmax = zsize;

    hipStream_t stream;
    T *u, *v, *u_host;
    std::ofstream os;

    check(hipHostMalloc(&u_host, xsize * ysize * zsize * sizeof(T)));
    host::initialise(u_host, xsize, ysize, zsize);

    os.open("in_field.csv");
    host::write_file(os, u_host, xsize, ysize, zsize);
    os.close();

    const time_point begin = std::chrono::steady_clock::now();
    check(hipStreamCreate(&stream));

    #if CUDART_VERSION >= 11020
    // Requires at least CUDA v11.2 to be installed.
    check(hipMallocAsync(&u, xsize * ysize * zsize * sizeof(T), stream));
    check(hipMallocAsync(&v, xsize * ysize * zsize * sizeof(T), stream));
    #else
    // Necessary for running on Piz Daint (as it only has CUDA v11.0 installed).
    check(hipMalloc(&u, xsize * ysize * zsize * sizeof(T)));
    check(hipMalloc(&v, xsize * ysize * zsize * sizeof(T)));
    #endif

    check(hipMemcpyAsync(u, u_host, xsize * ysize * zsize * sizeof(T), hipMemcpyHostToDevice, stream));

    switch(mode) {
        case Mode::laplap_global: {
            for(std::size_t i = 0; i < itrs; ++i) {
                device::update_boundaries(stream, u, xmin, xmax, ymin, ymax, zmax, xsize, ysize);
                device::update_interior_double_laplacian(stream, u, v, alpha, xmin, xmax, ymin, ymax, zmax, xsize, ysize);
            }
            break;
        }
        case Mode::laplap_shared: {
            for(std::size_t i = 0; i < itrs; ++i) {
                device::update_boundaries(stream, u, xmin, xmax, ymin, ymax, zmax, xsize, ysize);
                device::update_interior_double_laplacian_shared(stream, u, v, alpha, xmin, xmax, ymin, ymax, zmax, xsize, ysize);
            }
            break;
        }
        case Mode::biharm_global: {
            for(std::size_t i = 0; i < itrs; ++i) {
                device::update_boundaries(stream, u, xmin, xmax, ymin, ymax, zmax, xsize, ysize);
                device::update_interior_biharmonic(stream, u, v, alpha, xmin, xmax, ymin, ymax, zmax, xsize, ysize);
            }
            break;
        }
        default: __builtin_unreachable();
    }
    device::update_boundaries(stream, u, xmin, xmax, ymin, ymax, zmax, xsize, ysize);

    check(hipMemcpyAsync(u_host, u, xsize * ysize * zsize * sizeof(T), hipMemcpyDeviceToHost, stream));

    #if CUDART_VERSION >= 11020
    // Requires at least CUDA v11.2 to be installed.
    check(hipFreeAsync(u, stream));
    check(hipFreeAsync(v, stream));
    #else
    // Necessary for running on Piz Daint (as it only has CUDA v11.0 installed).
    check(hipFree(u));
    check(hipFree(v));
    #endif

    check(hipStreamDestroy(stream));
    check(hipDeviceSynchronize());
    const time_point end = std::chrono::steady_clock::now();

    os.open("out_field.csv");
    host::write_file(os, u_host, xsize, ysize, zsize);
    os.close();

    check(hipHostFree(u_host));

    return std::chrono::duration<double, std::milli>(end - begin).count() / 1000;
}


// templated_main<T>():
// Main function with flexible numeric real type.
//
// Input:   argv            :: Input arguments
//          argc            :: Number of input arguments
//          T               :: Numeric real type
// Output:  return (...)    :: Exit code (EXIT_SUCCESS or EXIT_FAILURE)
template<typename T>
int templated_main(int argc, char const **argv) {
    constexpr std::size_t bdry = 3;

    if(argc == 6) {
        std::size_t x, y, z, itrs;
        Mode mode;

        {
            std::istringstream x_ss(argv[1]), y_ss(argv[2]), z_ss(argv[3]), itrs_ss(argv[4]);
            x_ss >> x; y_ss >> y; z_ss >> z; itrs_ss >> itrs;
            mode = utils::mode_from_string(argv[5]);

            if(x_ss.fail() || y_ss.fail() || z_ss.fail() || itrs_ss.fail() ||
               x == 0 || y == 0 || z == 0 || itrs == 0 || mode == Mode::invalid) {

                utils::print_args_errmsg();
                return EXIT_FAILURE;
            }
        }

        std::cout << "================================================================================\n";
        std::cout << "                             Welcome to stencil2d!\n";
        std::cout << "Version    :: C++ with CUDA v" << CUDART_VERSION / 1000 << '.' << CUDART_VERSION / 10 % 100 << '\n';
        std::cout << "Interior   :: (" << x << ", " << y << ", " << z << ")\n";
        std::cout << "Boundaries :: (" << bdry << ", " << bdry << ", " << 0 << ")\n";
        std::cout << "Iterations :: " << itrs << '\n';
        std::cout << "Real size  :: " << sizeof(T) << '\n';
        std::cout << "Exec. mode :: " << utils::get_mode_desc(mode) << '\n';
        std::cout << "================================================================================\n";

        const double time = run_simulation<T>(x + 2 * bdry, y + 2 * bdry, z, itrs, bdry, mode);

        std::cout << "Runtime    :: " << time << "s\n";
        std::cout << "================================================================================\n";
    }
    else {
        utils::print_args_errmsg();
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}


int main(int argc, char const **argv) {
    #if !defined(REALSIZE) || REALSIZE == 8
    return templated_main<double>(argc, argv);
    #elif REALSIZE == 4
    return templated_main<float>(argc, argv);
    #else
    #error "Selected REALSIZE not supported!"
    #endif
}
